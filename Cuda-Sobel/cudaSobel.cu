#include "hip/hip_runtime.h"
﻿
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <opencv2/opencv.hpp>
#include "iostream"
#define THREAD_DIM 20 // Cuda threadlimits in one block: 32 * 32 = 1024
#define NANO_TO_MILLI 1e-6

using namespace cv;
using namespace std;

/**
 * @brief Horizontal Sobel as parallel implementation on GPU. Version for Cuda with 2-Dim grid and 2-Dim blocks.
 *
 * @param image black-white image
 * @param result result image after horizental filtering
 * @param height height of image
 * @param width width of image
 */
__global__ void horizontalSobel(const uchar *image, uchar *result, const int height, const int width) {
    const int row = threadIdx.x + blockIdx.x * blockDim.x;
    const int col = threadIdx.y + blockIdx.y * blockDim.y;

    if(row <= height - 2 && col <= width - 2){
        uchar xDerivate = image[row * width + col] - image[row * width + col + 2]
	        + 2 * image[(row + 1) * width + col] - 2 * image[(row + 1) * width + col + 2]
	        + image[(row + 2) * width + col] - image[(row + 2) * width + col + 2];

        result[row * width + col] = xDerivate;
    }
}

/**
 * @brief Convert OpenCV matrix to uchar array.
 *
 * @param matrix data of image
 * @param array result images pixels in an array
 * @param rows height of image
 * @param cols width of image
 */
void matrixToArray(const Mat matrix, uchar *array, const int rows, const int cols){
    for (int row = 0; row < rows; row++) {
        for (int col = 0; col < cols; col++) {
            array[row * cols + col] = matrix.at<uchar>(row, col);
        }
    }
}

/**
 * @brief Update OpenCV matrix with data of uchar array.
 *
 * @param array images pixels in an array
 * @param matrix data of image to update
 * @param rows height of image
 * @param cols width of image
 */
void arrayToMatrix(const uchar *array, Mat matrix, const int rows, const int cols){
    for (auto row = 0; row < rows; row++) {
        for (auto col = 0; col < cols; col++) {
            matrix.at<uchar>(row, col) = array[row * cols + col];
        }
    }
}

int main(void) {
    // Read the image file
    const string IMAGE_PATH = "../images/";
    const string IMAGE_DIMENSION = "4500";

    string imageName = IMAGE_PATH + "horses_" + IMAGE_DIMENSION + ".jpg";
    Mat image = imread(imageName, IMREAD_GRAYSCALE);

    // Check for failure
    if (image.empty()) {
        cout << "Image Not Found!!!" << endl;
        cin.get(); //wait for any key press
        return -1;
    }

    // Prepare and convete images data
	const int PIXELS = image.rows * image.cols;

	auto *imageArray = (uchar *)malloc(PIXELS * sizeof(char));
	auto *imageResultArray = (uchar *)malloc(PIXELS * sizeof(char));
    matrixToArray(image, imageArray, image.rows, image.cols);

    // Cuda stuff
    dim3 threads(THREAD_DIM, THREAD_DIM, 1);
    dim3 blocks(ceil(image.rows/(double)THREAD_DIM), ceil(image.cols/(double)THREAD_DIM), 1);
    printf("Blocks: %d, threads per block: %d", blocks.x * blocks.y, threads.x * threads.y);

    uchar *devImageArray;
    uchar *devImageResultArray;
    hipMalloc((void**)&devImageArray, PIXELS * sizeof(char));
    hipMalloc((void**)&devImageResultArray, PIXELS * sizeof(char));

    hipMemcpy(devImageArray, imageArray, PIXELS * sizeof(char), hipMemcpyHostToDevice);

    auto begin = std::chrono::high_resolution_clock::now();
    horizontalSobel<<<blocks, threads>>>(devImageArray, devImageResultArray, image.rows, image.cols);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
	auto execTime = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);

    hipMemcpy(imageResultArray, devImageResultArray, PIXELS * sizeof(char), hipMemcpyDeviceToHost);
    arrayToMatrix(imageResultArray, image, image.rows, image.cols);

    hipFree(devImageArray);
    hipFree(devImageResultArray);
	free(imageArray);
	free(imageResultArray);

    string imageResultName = IMAGE_PATH + "horses_" + IMAGE_DIMENSION + "_sobel.jpg";
    imwrite(imageResultName, image);

    image.release();

    double execTimeSobel = ((double)execTime.count() * NANO_TO_MILLI);
	printf("\nExect time: %f ms\n", execTimeSobel);
}